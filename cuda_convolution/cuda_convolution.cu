#include <stdio.h>
#include<iostream>
#include <cmath> 
#include <hip/hip_runtime.h>
#include <iomanip>
#include <chrono>
#include <cstring>

#define DEBUG 0

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void print_time_elapsed(std::chrono::time_point<std::chrono::high_resolution_clock> start, std::chrono::time_point<std::chrono::high_resolution_clock> end, 
                        int filter_size, int width, int height, int nb_iters){
    double mega_pixels = (width*height*nb_iters)/1e6;
    double flop = ((filter_size*filter_size) + (filter_size*filter_size - 1))*mega_pixels;                            
    std::chrono::duration<double> elapsed_seconds = end - start;
    double seconds = elapsed_seconds.count();
    double flops = (flop)/(seconds*1e3);
    std::cout << "Time taken: " << seconds  <<  " seconds" << std::endl;
    std::cout << "GFlops: " << flops << std::endl;
    std::cerr << seconds << std::endl;
}

__global__ void convolve(float *image, float *res_image, float *filter, int height, int width, int filter_size, int padded_width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < height*width){
        float res = 0;
        int y = int(idx/width), x = idx%width;
        for(int i=y; i<y+filter_size; i++){
            for(int j=x; j<x+filter_size; j++){
                res += image[i*padded_width + j]*filter[(i-y)*filter_size + (j-x)];
            }
        }
        res_image[idx] = res;
    }
}

void init_image(float *image, int width, int height, int padding){
    long int pixel_val = 1;
    for (long int y = 0; y < height; y++) {
        for (long int x = 0; x < width; x++) {
            // Account for padding.
            if ((y<padding)|(x<padding)|(y>=(height-padding))|(x>=(width-padding))){
                image[y*width + x] = 0;
                continue;   
            }
            else{
                    image[y*width + x] = pixel_val;
                    pixel_val++;
            }
        }
    }
}

void generate_identity_kernel(float *&filter, int filter_size){
    for(int i=0; i<filter_size; i++){
        for(int j=0; j<filter_size; j++){
            filter[i*filter_size + j] = 0;
        }
    }
    int mid_point = int(filter_size/2);
    filter[mid_point*filter_size + mid_point] = 1;
}

void print_image(float *image, int width, int height){
    for (int y=0; y<height; y++){
        for (int x=0; x<width; x++){
            std::cout << std::setw(5) << std::setfill(' ') << image[y*width + x] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "\n";
}

void validate_result(float *result, long int size){
    for(long int i=0; i<size; i++){
        if (result[i] != i+1){
            std::cerr << "Wrong answer at index: " << i-1 << std::endl;
            exit(0);
        }
    }
    std::cout << "All pixels checked and verified!\n";
}


int main(int argc, char *argv[]) {
    int height = atoi(argv[1]);
    int width = atoi(argv[2]);
    int filter_size = atoi(argv[3]);

    int padding = 2*int(filter_size/2);

    float* h_image = new float[(height+padding)*(width+padding)];
    float* h_result = new float[height*width];
    float* h_filter = new float[filter_size*filter_size];

    // std::cout << "Ivide" << h_image[(height+padding)*(width+padding)-1] << "\n";
    init_image(h_image, width+padding, height+padding, padding/2);
    generate_identity_kernel(h_filter, filter_size);

    #if DEBUG
        print_image(h_filter, filter_size, filter_size);
        print_image(h_image, width+padding, height+padding);
    #endif

    float *d_image, *d_filter, *d_result;
    gpuErrchk(hipMalloc(&d_image, (height+padding)*(width+padding)*sizeof(float)));
    gpuErrchk(hipMalloc(&d_filter, filter_size*filter_size*sizeof(float)));
    gpuErrchk(hipMalloc(&d_result, height*width*sizeof(float)));
    
    gpuErrchk(hipMemcpy(d_image, h_image, (height+padding)*(width+padding)*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_filter, h_filter, filter_size*filter_size*sizeof(float), hipMemcpyHostToDevice));

    int threadsPerBlock = 512;
    int blocksPerGrid = (height*width + threadsPerBlock - 1) / threadsPerBlock;
    
    auto start_compute = std::chrono::high_resolution_clock::now();
    
    int nb_iters = 10;
    for(int i=0; i<nb_iters; i++){
        convolve<<<blocksPerGrid, threadsPerBlock>>>(d_image, d_result, d_filter, height, width, filter_size, width+padding);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError() );
    }
    
    auto end_compute = std::chrono::high_resolution_clock::now();
    
    gpuErrchk(hipMemcpy(h_result, d_result, height*width*sizeof(float), hipMemcpyDeviceToHost));

    #if DEBUG
        print_image(h_result, width, height);
    #endif
    
    validate_result(h_result, height*width);

    print_time_elapsed(start_compute, end_compute, filter_size, width, height, nb_iters);
    
    gpuErrchk(hipFree(d_image));
    gpuErrchk(hipFree(d_filter));
    gpuErrchk(hipFree(d_result));

    delete[] h_image;
    delete[] h_result;
    delete[] h_filter;
    
    return 0;
}